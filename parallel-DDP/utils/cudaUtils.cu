/*****************************************************************
 * Utils for Cuda code
 * 1 CUDA ERROR CHECKING CODE
 *
 * Adapted from:
 * https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
 * https://stackoverflow.com/questions/22399794/qr-decomposition-to-solve-linear-systems-in-cuda
 *
 * error checking usage for library functions:
 *    gpuErrchk(cudaMalloc((void**)&a_d, size*sizeof(int)));
 *    cusolveErrchk(<cusolvefunctioncall>)
 *    cublasErrchk(<cubalsfunctioncall>)
 * error checking usage for custom kernels:
 *    kernel<<<1,1>>>(a);
 *    gpuErrchk(cudaPeekAtLastError());
 *    gpuErrchk(cudaDeviceSynchronize());
 *
 * 2 Matrix printing code
 *
 * 3 Templated External Memory Wrapper
 *****************************************************************/
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hipsolver.h>


/*** 1 CUDA ERROR CHECKING CODE 1 ***/
    __host__ 
    void gpuAssert(hipError_t code, const char *file, const int line, bool abort=true){
        if (code != hipSuccess){
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort){hipDeviceReset(); exit(code);}
        }
    }

    __host__ __device__ 
    static const char *cublasGetErrorEnum(hipblasStatus_t error){
        switch (error){
            case HIPBLAS_STATUS_SUCCESS:             return "CUBLAS_STATUS_SUCCESS";
            case HIPBLAS_STATUS_NOT_INITIALIZED:     return "CUBLAS_STATUS_NOT_INITIALIZED";
            case HIPBLAS_STATUS_ALLOC_FAILED:        return "CUBLAS_STATUS_ALLOC_FAILED";
            case HIPBLAS_STATUS_INVALID_VALUE:       return "CUBLAS_STATUS_INVALID_VALUE";
            case HIPBLAS_STATUS_ARCH_MISMATCH:       return "CUBLAS_STATUS_ARCH_MISMATCH";
            case HIPBLAS_STATUS_MAPPING_ERROR:       return "CUBLAS_STATUS_MAPPING_ERROR";
            case HIPBLAS_STATUS_EXECUTION_FAILED:    return "CUBLAS_STATUS_EXECUTION_FAILED";
            case HIPBLAS_STATUS_INTERNAL_ERROR:      return "CUBLAS_STATUS_INTERNAL_ERROR";
            case HIPBLAS_STATUS_NOT_SUPPORTED:       return "CUBLAS_STATUS_NOT_SUPPORTED";
            case HIPBLAS_STATUS_UNKNOWN:       return "CUBLAS_STATUS_LICENSE_ERROR";
            default:                                return "<unknown>";
        }
    }

    __host__ __device__ 
    void cublasAssert(hipblasStatus_t err, const char *file, const int line){
        if(HIPBLAS_STATUS_SUCCESS != err) {
            #ifdef  __CUDA_ARCH__
                printf("CUBLAS error in file '%s', line %d\n error %d: %s\n terminating!\n",file,line,err,cublasGetErrorEnum(err));
            #else
                fprintf(stderr, "CUBLAS error in file '%s', line %d\n error %d: %s\n terminating!\n",file,line,err,cublasGetErrorEnum(err));
                hipDeviceReset();
            #endif
            assert(0);
        }
    }

    __host__ 
    static const char *cusolverGetErrorEnum(hipsolverStatus_t error){
        switch (error){
            case HIPSOLVER_STATUS_SUCCESS:                   return "CUSOLVER_SUCCESS";
            case HIPSOLVER_STATUS_NOT_INITIALIZED:           return "CUSOLVER_STATUS_NOT_INITIALIZED";
            case HIPSOLVER_STATUS_ALLOC_FAILED:              return "CUSOLVER_STATUS_ALLOC_FAILED";
            case HIPSOLVER_STATUS_INVALID_VALUE:             return "CUSOLVER_STATUS_INVALID_VALUE";
            case HIPSOLVER_STATUS_ARCH_MISMATCH:             return "CUSOLVER_STATUS_ARCH_MISMATCH";
            case HIPSOLVER_STATUS_EXECUTION_FAILED:          return "CUSOLVER_STATUS_EXECUTION_FAILED";
            case HIPSOLVER_STATUS_INTERNAL_ERROR:            return "CUSOLVER_STATUS_INTERNAL_ERROR";
            case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "CUSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
            default:                                        return "<unknown>";
        }
    }

    __host__ void cusolverAssert(hipsolverStatus_t err, const char *file, const int line){
        if(HIPSOLVER_STATUS_SUCCESS != err) {
            fprintf(stderr, "CUSOLVE error in file '%s', line %d\n error %d: %s\n terminating!\n",file,line,err,cusolverGetErrorEnum(err));
            hipDeviceReset(); assert(0);
        }
    }

    __host__ 
    static const char *cusparseGetErrorEnum(hipsparseStatus_t error){
        switch (error){
            case HIPSPARSE_STATUS_SUCCESS:                   return "CUSPARSE_SUCCESS";
            case HIPSPARSE_STATUS_NOT_INITIALIZED:           return "CUSPARSE_STATUS_NOT_INITIALIZED";
            case HIPSPARSE_STATUS_ALLOC_FAILED:              return "CUSPARSE_STATUS_ALLOC_FAILED";
            case HIPSPARSE_STATUS_INVALID_VALUE:             return "CUSPARSE_STATUS_INVALID_VALUE";
            case HIPSPARSE_STATUS_ARCH_MISMATCH:             return "CUSPARSE_STATUS_ARCH_MISMATCH";
            case HIPSPARSE_STATUS_EXECUTION_FAILED:          return "CUSPARSE_STATUS_EXECUTION_FAILED";
            case HIPSPARSE_STATUS_INTERNAL_ERROR:            return "CUSPARSE_STATUS_INTERNAL_ERROR";
            case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "CUSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
            default:                                        return "<unknown>";
        }
    }

    __host__ void cusparseAssert(hipsparseStatus_t err, const char *file, const int line){
        if(HIPSPARSE_STATUS_SUCCESS != err) {
            fprintf(stderr, "CUSPARSE error in file '%s', line %d\n error %d: %s\n terminating!\n",file,line,err,cusparseGetErrorEnum(err));
            hipDeviceReset(); assert(0);
        }
    }
  
/*** 1 CUDA ERROR CHECKING CODE 1 ***/