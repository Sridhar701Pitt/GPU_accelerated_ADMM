#include "hip/hip_runtime.h"
/***
nvcc -std=c++11 -o MPC.exe WAFR_MPC_examples.cu utils/cudaUtils.cu utils/threadUtils.cpp -llcm -gencode arch=compute_61,code=sm_61 -rdc=true -O3
***/
#define EE_COST 1
#define MPC_MODE 1
#define IGNORE_MAX_ROX_EXIT 0
#define TOL_COST 0.00001
#define PLANT 4
#include "DDPHelpers.cuh"
#include <random>
#include <vector>
#include <algorithm>
#include <iostream>

#define TEST_ITERS 1 // 100
#define ROLLOUT_FLAG 0
#define RANDOM_MEAN 0.0
#define RANDOM_STDEV 0.001
char errMsg[]  = "Error: Unkown code - usage is [C]PU or [G]PU with flag 1/0 for doFig8\n";
char tot[]  = "  TOT";		char init[] = " INIT";	char fsim[]   = "  SIM";	
char fsweep[]   = "SWEEP";	char bp[]   = "   BP";	char nis[]  = "  NIS";
std::default_random_engine randEng(time(0)); //seed
std::normal_distribution<double> randDist(RANDOM_MEAN, RANDOM_STDEV); //mean followed by stdiv

#if PLANT == 1 // pend
	#error "MPC example defined for KukaArm[4].\n"
#elif PLANT == 2 // cart
	#error "MPC example defined for KukaArm[4].\n"
#elif PLANT == 3 // quad
 	#error "MPC example defined for KukaArm[4].\n"
#elif PLANT == 4 // arm
 	#if EE_COST
		#define PI 	   3.14159
		#define GOAL_X 0.3638
		#define GOAL_Y 0.0
		#define GOAL_Z 1.0628
		#define GOAL_r (0.5*PI)
		#define GOAL_p 0.0
		#define GOAL_y (0.5*PI)
	#else  
		#error "MPC example defined for KukaArm[4] with EE_COST.\n"
	#endif
#else
	#error "MPC example defined for KukaArm[4].\n"
#endif

__host__ __forceinline__
bool tryParse(std::string& input, int& output) {
	try{output = std::stoi(input);}
	catch (std::invalid_argument) {return false;}
	return true;
}
__host__ __forceinline__
int getInt(int maxInt, int minInt){
	std::string input;	std::string exitCode ("q"); int x;
	while(1){
		getline(std::cin, input);
		while (!tryParse(input, x)){
			if (input.compare(input.size()-1,1,exitCode) == 0){return -1;}
				std::cout << "Bad entry. Enter a NUMBER\n";	getline(std::cin, input);
			}
		if (x >= minInt && x <= maxInt){break;}
		else{std::cout << "Entry must be in range[" << minInt << "," << maxInt << "]\n";}
	}
	return x;
}
__host__ __forceinline__
int getTrajTime(int maxInt, int minInt){
   printf("How many seconds long should the tracked trajectory be? (q to exit)\n");
   return getInt(maxInt,minInt);
}
__host__ __forceinline__
int getTimeBudget(int maxInt, int minInt){
   printf("What should the MPC time budget be (in ms)? (q to exit)?\n");
   return getInt(maxInt,minInt);
}
__host__ __forceinline__
int getMaxIters(int maxInt, int minInt){
   printf("What is the maximum number of iterations a solver can take? (q to exit)?\n");
   return getInt(maxInt,minInt);
}
__host__ __forceinline__
void keyboardHold(){
   	printf("Press enter to continue\n");	std::string input;	getline(std::cin, input);
}
__host__
void printStats(std::vector<double> v, char *type){
	// sort gives us the median, max and min
    std::sort(v.begin(),v.end());
    int size = v.size();
	double _median = size % 2 ? v[size / 2] : (v[size / 2 - 1] + v[size / 2]) / 2.0;	
	double _max = v.back();		double _min = v.front();	double _stdev = 0.0;
	// sum gives us the average
	double sum = std::accumulate(v.begin(), v.end(), 0.0);	double _avg = sum / (double)size;	
	// and then the std dev	
	for(std::vector<double>::iterator it = v.begin(); it != v.end(); ++it){_stdev += pow(*it-_avg,2.0);}
	_stdev = pow(_stdev / (double)size, 0.5);
	printf("%s: Median[%f] Average[%f] StdDev[%f] max[%f] min[%f]\n",type,_median,_avg,_stdev,_max,_min);
}
template <typename T>
__host__ 
void printAllTimingStats(algTrace<T> *atrace){
	printStats(atrace->tTime,tot);
	printStats(atrace->initTime,init);
	printStats(atrace->simTime,fsim);
	printStats(atrace->sweepTime,fsweep);
	printStats(atrace->bpTime,bp);
	printStats(atrace->nisTime,nis);
}
template <typename T>
__host__ __forceinline__
void loadTraj(trajVars<T> *tvars, matDimms *dimms){
	T *xk = tvars->x;	T *uk = tvars->u;
	// for (int k=0; k<NUM_TIME_STEPS; k++){
	// 	xk[0] = (T)-0.5*PI;	xk[1] = (T)0.25*PI;	xk[2] = (T)0.167*PI;
	// 	xk[3] = (T)-0.167*PI;	xk[4] = (T)0.125*PI;	xk[5] = (T)0.167*PI;	xk[6] = (T)0.5*PI;
	// 	xk[7] = (T)randDist(randEng);	xk[8] = (T)randDist(randEng);	xk[9] = (T)randDist(randEng);
	// 	xk[10] = (T)randDist(randEng);	xk[11] = (T)randDist(randEng);	xk[12] = (T)randDist(randEng);	xk[13] = (T)randDist(randEng);
	// 	if (k < NUM_TIME_STEPS - 1){
	// 		uk[0] = 0.0;		uk[1] = -102.9832;	uk[2] = 11.1968;
	// 		uk[3] = 47.0724;	uk[4] = 2.5993;		uk[5] = -7.0290;	uk[6] = -0.0907;
	// 		//uk[0] = (T)0.01; uk[1] = (T)0.01; uk[2] = (T)0.01; uk[3] = (T)0.01; uk[4] = (T)0.01; uk[5] = (T)0.01; uk[6] = (T)0.01;
	// 	}
	// 	xk += (dimms->ld_x);	uk += (dimms->ld_u);
	// }
	for (int k=0; k<NUM_TIME_STEPS; k++){
		for (int i = 0; i < STATE_SIZE; i++){
			xk[i] = 0.0;	if (i < CONTROL_SIZE){uk[i] = 0.01;}
		}
		xk += (dimms->ld_x);	uk += (dimms->ld_u);
	}
	// memset(tvars->x, 0, (dimms->ld_x)*NUM_TIME_STEPS*sizeof(T));
	// memset(tvars->u, 0, (dimms->ld_u)*NUM_TIME_STEPS*sizeof(T));
	memset(tvars->KT, 0, (dimms->ld_KT)*DIM_KT_c*NUM_TIME_STEPS*sizeof(T));
}
template <typename T>
__host__ __forceinline__
void loadGoal(CPUVars<T> *algvars){
	const T temp[] = {GOAL_X,GOAL_Y,GOAL_Z,GOAL_r,GOAL_p,GOAL_y};
	for (int i=0; i < 6; i++){(algvars->xGoal)[i] = temp[i];}
}
template <typename T>
__host__ __forceinline__
void loadGoal(GPUVars<T> *algvars){
	const T temp[] = {GOAL_X,GOAL_Y,GOAL_Z,GOAL_r,GOAL_p,GOAL_y};
	for (int i=0; i < 6; i++){(algvars->xGoal)[i] = temp[i];}
}
template <typename T>
__host__ __forceinline__
int loadGoal(T *goal, T *xGoals, T *yGoals, T *zGoals, double time, double tstep, double totalTime){
	int rep = 0;
	while(time > totalTime){time -= totalTime; rep++;}
	double delta = time/tstep;							double fraction = delta - std::floor(delta);
	int rd = static_cast<int>(std::floor(delta));		int ru = static_cast<int>(std::ceil(delta));
	goal[0] = (1-fraction)*xGoals[rd] + fraction*xGoals[ru];
	goal[1] = (1-fraction)*yGoals[rd] + fraction*yGoals[ru];
	goal[2] = (1-fraction)*zGoals[rd] + fraction*zGoals[ru];
	return rep;
}
template <typename T>
__host__ __forceinline__
int loadGoal(CPUVars<T> *algvars, T *xGoals, T *yGoals, T *zGoals, double time, double tstep, double totalTime){
	T goal[3];
	int rep = loadGoal(&goal[0],xGoals,yGoals,zGoals,time,tstep,totalTime);
	algvars->xGoal[0] = goal[0];	algvars->xGoal[1] = goal[1];	algvars->xGoal[2] = goal[2];
	algvars->xGoal[3] = 0;			algvars->xGoal[4] = 0;			algvars->xGoal[5] = 0;
	return rep;
}
template <typename T>
__host__ __forceinline__
int loadGoal(GPUVars<T> *algvars, T *xGoals, T *yGoals, T *zGoals, double time, double tstep, double totalTime){
	T goal[3];
	int rep = loadGoal(&goal[0],xGoals,yGoals,zGoals,time,tstep,totalTime);
	algvars->xGoal[0] = goal[0];	algvars->xGoal[1] = goal[1];	algvars->xGoal[2] = goal[2];
	algvars->xGoal[3] = 0;			algvars->xGoal[4] = 0;			algvars->xGoal[5] = 0;
	return rep;
}
template <typename T>
__host__ __forceinline__
int loadFig8Goal(T *goal, double time, double totalTime){
	T xGoals[] = {0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004,0.6556285000000004};
	T yGoals[] = {0.13686922001827645,0.1281183229143938,0.11926059413541247,0.11030432312999372,0.1012577993467988,0.09212931223448896,0.08292715124172548,0.0736596058171696,0.06433496540948258,0.05496151946732569,0.045547557439360176,0.03610136877424733,0.026631242920648335,0.017145469327224508,0.007652337442637103,-0.001839863284452653,-0.011322843405383448,-0.020788313471494096,-0.030227984034123273,-0.039633565644609764,-0.0489967688542923,-0.05830930421450961,-0.06756288227660043,-0.07674921359190354,-0.08586000871175764,-0.09488697818749958,-0.1038655258329727,-0.11272375591745916,-0.12145267616480321,-0.13004109492853494,-0.13848752180117063,-0.1467858524541864,-0.1549281455978588,-0.1629165235905652,-0.1707482796892309,-0.17841599130217783,-0.18591882035832843,-0.1932502389635062,-0.20040076105913676,-0.20735798772338723,-0.21411519853223893,-0.22066551450768562,-0.2269966386554578,-0.2331053523296066,-0.23897989082424964,-0.24461177603116246,-0.24999586174091798,-0.255127737887603,-0.2600037985005273,-0.264623973362925,-0.2689800086072113,-0.27306787696210033,-0.2768840041676995,-0.2804279779836409,-0.28369462501981374,-0.28667669709392635,-0.2893761036490012,-0.29178950599107006,-0.29391158123626243,-0.29574034972478985,-0.29727453396452347,-0.29851358128156347,-0.2994544451742904,-0.30009482660988346,-0.30043907902783173,-0.30048644363405164,-0.3002369544043975,-0.29968819678814346,-0.2988444313574429,-0.2977048064508765,-0.2962719042333858,-0.29454317198576974,-0.2925200231035754,-0.2902045731500091,-0.2876022809123594,-0.2847133796914938,-0.28154140071122247,-0.27809048676073667,-0.27436270583704647,-0.2703665745543449,-0.26610524813698555,-0.26158238012185503,-0.25680474076422527,-0.2517742032662073,-0.2464935808285531,-0.24096598110942027,-0.23519855715870805,-0.22919065210989617,-0.22294815964121067,-0.216477551020404,-0.20978904844956445,-0.20289009717093992,-0.19578529751522875,-0.18849136560284127,-0.1810102720333381,-0.17335332623615154,-0.16552932116242403,-0.15754446533330008,-0.1494055335341655,-0.141119776212369,-0.13269158757842503,-0.12412621123921029,-0.11543344583116043,-0.10661783052843572,-0.09768974155699829,-0.08866263788534111,-0.07955032956578706,-0.07036309089673469,-0.06111613853088287,-0.051824258760106816,-0.04250384833178472,-0.03316965050170948,-0.02383739609051122,-0.014517898480146497,-0.00521285872839692,0.004084069854356758,0.013377716369566438,0.022678964415954628,0.03199791841264746,0.04134623887290685,0.05072829056150665,0.06014998301207159,0.06960936374549687,0.07908892160637689,0.08855068744478754,0.09794356384062872,0.10720878615596977,0.11629326145552335,0.12514991959047528,0.13374545516319478,0.14207020091055017,0.15012393938707272,0.15792746608076888,0.16551126865840357,0.1729042968130615,0.18013571994835326,0.1872239892183252,0.19418299168723718,0.20100573965434845,0.2076839316858779,0.2142059797503866,0.22054968628960595,0.22669845070103078,0.2326348093954518,0.2383522458039774,0.24384287073319427,0.2491022174907651,0.2541293098371375,0.2589202699947854,0.2634683865999179,0.2677700650071298,0.27181899023759926,0.2756002672767167,0.2791048721375289,0.28231577838682104,0.28522354073704403,0.2878230537497963,0.2901076694828819,0.2920783437903407,0.29374310403406084,0.295107993386028,0.2961895263765827,0.2970056784978085,0.2975819271104611,0.29794495005548816,0.298109264082986,0.2980740741579594,0.2978323971097816,0.2973591836734637,0.296622924641549,0.2955794967043364,0.29418154658089085,0.2923728011959445,0.2900962060295761,0.2873037962354698,0.28396753800504204,0.28032650994085717,0.2763886650341475,0.27216195627614015,0.26765433665806226,0.2628737591711411,0.2578281768066035,0.25252554255567694,0.2469738094095883,0.2411809303595648,0.2351548583968336,0.2289035465126218,0.22243494769815644,0.21575701494466482,0.208877701243374,0.201804959585511,0.19454674296230312,0.1871110043649775,0.179505696784761,0.1717387732128811,0.16381818664056474,0.1557518900590391,0.14754783645953137,0.1392139788332685,0.1307582701714778};
	T zGoals[] = {0.31756840547539744,0.32748579299353975,0.3378005717933035,0.3484792783644661,0.3594884491968049,0.3707946207800972,0.38236432960412026,0.39416411215865155,0.40616050493346834,0.4183200444183479,0.4306092671030677,0.44299470947740494,0.455442908031137,0.4679203992540413,0.48039371963589506,0.49282940566647565,0.5051939938355604,0.5174540206329267,0.5295760225483518,0.541526536071613,0.5532720976924878,0.5647792439007534,0.5760145111861872,0.5869444360385665,0.5975355549476685,0.607754404403259,0.617684862624283,0.6269833110602612,0.6356398015809976,0.6436439919363848,0.6509877754875409,0.6576613603932012,0.6636610682008525,0.6689826360732843,0.67361769235995,0.6775587102765505,0.6807894425694675,0.6832982853518634,0.6850685995152646,0.6860878343790213,0.6863475050397387,0.6858378423067206,0.6845542073433015,0.6824970237151333,0.679674950734998,0.6760903644476528,0.6717601780334646,0.6666986364356934,0.6609289904640975,0.6544774785385726,0.6473731907857357,0.6396420054814256,0.6313175949625012,0.6224353167682047,0.6130293234274724,0.6031408593626014,0.592806201725962,0.5820708170060365,0.5709740379170426,0.5595578699404182,0.5478698928627949,0.5359537973679934,0.5238581689279385,0.5116302430406027,0.4993208702348768,0.48697884889804044,0.4746532265736393,0.4623956012593628,0.4502485492310045,0.4382596910462214,0.42647866542842733,0.41494764320821875,0.4037154159777041,0.39282851027202587,0.3823335341683771,0.37227688282859595,0.36269856168881953,0.3536372194160615,0.3451304536909037,0.33721132075471055,0.32990808625336293,0.32324779496703726,0.3172587080115004,0.31196097533352374,0.3073723368592661,0.3035111825862362,0.3003931753833552,0.29803437903462704,0.2964420439874006,0.29562574916758594,0.2955914743255737,0.29633569503272444,0.29786307504133164,0.3001661781693752,0.30323465990169074,0.3070520940451594,0.3115998595664622,0.31685456748736623,0.3227967201975068,0.32939373258736826,0.336620401368088,0.34444137126548074,0.35282689981652365,0.3617486647489115,0.37117885795827055,0.3810878162585716,0.39144908831456365,0.4022280096944361,0.41338530657545247,0.42488278556704895,0.43667841400710394,0.4487240639652002,0.46096913859882693,0.47336419850958344,0.48584480962195276,0.4983474665337738,0.5108078008561817,0.5231640943169621,0.5353543092708679,0.5473260617454424,0.5590277695983947,0.5704169154454125,0.5814512174681081,0.5920942512854084,0.6023199392964617,0.6120999705542476,0.6214094905886763,0.6302189177557694,0.6384948605857313,0.6462095585403849,0.6533240617001391,0.6598152959296799,0.6656559786178493,0.6708180864798622,0.6752789417653723,0.6790134023420558,0.6819951301275096,0.6841972139799184,0.6856012933475145,0.6861863032005122,0.6859428344922742,0.6848659833744687,0.6829691340687087,0.6802671521438632,0.6767869940428872,0.6725559197263683,0.6675988289655357,0.6619506036376687,0.6556389703050904,0.6486908922059266,0.6411394935332501,0.6330153548211616,0.6243554553896911,0.6151955559582207,0.6055661079526033,0.5954998822170263,0.5850395750753018,0.5742180207932119,0.563083467349193,0.5516783030193103,0.5400466850891197,0.5282336225055959,0.5162846587691582,0.5042482411832561,0.49216898910506657,0.4800977462683095,0.4680759949262634,0.45614115835012375,0.4443280391401759,0.4326669709393109,0.42118572108087715,0.40991017010577024,0.3988536909103188,0.3880358943577357,0.37746325851094437,0.3677153816185399,0.3583626678734707,0.34943548878219216,0.34096421585115233,0.33297922058679913,0.3255108744955808,0.3185895490839454,0.31224561585834093,0.30650944632521565,0.30141141199101745,0.2969818843621946,0.29325123494519506,0.29024983524646697,0.28800805677245844,0.2865562710296175,0.28592484952439223,0.2861441637632308,0.28724458525258123,0.2892564854988917,0.2922102360086102,0.2961362082881848,0.30106477384406377,0.30702630418269494,0.3140511708105266,0.32216974523400665};
	int numGoals = 200; 	double tstep = totalTime/(numGoals-1);	double goalNum = time/tstep;
	double fraction = goalNum - std::floor(goalNum);				int rep = static_cast<int>(std::floor(goalNum)) / numGoals;
	int rd = static_cast<int>(std::floor(goalNum)) % numGoals;		int ru = static_cast<int>(std::ceil(goalNum)) % numGoals;
	goal[0] = (1-fraction)*xGoals[rd] + fraction*xGoals[ru];
	goal[1] = (1-fraction)*yGoals[rd] + fraction*yGoals[ru];
	goal[2] = (1-fraction)*zGoals[rd] + fraction*zGoals[ru];
	return rep;
}
template <typename T>
__host__ __forceinline__
int loadFig8Goal(CPUVars<T> *algvars, double time, double totalTime){
	T goal[3];		int rep = loadFig8Goal(&goal[0],time,totalTime);
	algvars->xGoal[0] = goal[0];	algvars->xGoal[1] = goal[1];	algvars->xGoal[2] = goal[2];
	algvars->xGoal[3] = 0;			algvars->xGoal[4] = 0;			algvars->xGoal[5] = 0;
	return rep;
}
template <typename T>
__host__ __forceinline__
int loadFig8Goal(GPUVars<T> *algvars, double time, double totalTime){
	T goal[3];		int rep = loadFig8Goal(&goal[0],time,totalTime);
	algvars->xGoal[0] = goal[0];	algvars->xGoal[1] = goal[1];	algvars->xGoal[2] = goal[2];
	algvars->xGoal[3] = 0;			algvars->xGoal[4] = 0;			algvars->xGoal[5] = 0;
	return rep;
}	
template <typename T>
__host__
void evNorm(T *xActual, T *xGoal, T *eNorm, T *vNorm){
	T eePos[NUM_POS];   compute_eePos_scratch<T>(xActual, &eePos[0]);
	*eNorm = static_cast<T>(sqrt(pow(eePos[0]-xGoal[0],2) + pow(eePos[1]-xGoal[1],2) + pow(eePos[2]-xGoal[2],2)));
	*vNorm = 0; for(int i=0;i<NUM_POS;i++){*vNorm+=(T)pow(xActual[NUM_POS+i],2);} *vNorm = static_cast<T>(sqrt(*vNorm));
}
template <typename T>
__host__
void evNorm(double *xActual, T *xGoal, T *eNorm, T *vNorm){
	T xActual2[STATE_SIZE];
	for(int i=0; i < STATE_SIZE; i++){xActual2[i] = static_cast<T>(xActual[i]);}
	evNorm<T>(xActual2,xGoal,eNorm,vNorm);
}
template <typename T, int SUBSTEPS>
__host__
T simulateForward(trajVars<T> *tvars, T *xActual, double elapsedTime, double goalTime, double totalTime){
	double I[36*NUM_POS]; double Tbody[36*NUM_POS]; initI<double>(I); initT<double>(Tbody);
	// and break elapsed time into SUBSTEPS for accuracy
	double qdes[NUM_POS], udes[CONTROL_SIZE], currX[STATE_SIZE], nextX[STATE_SIZE], qdd[NUM_POS];
	double dt_us = elapsedTime/static_cast<double>(SUBSTEPS); double dt = dt_us / 1000000.0;
	for(int j=0; j < STATE_SIZE; j++){currX[j] = static_cast<double>(xActual[j]);}
	double t0 = static_cast<double>(tvars->t0_plant); double tk = t0;
	T totalError = 0;	T goal[3];	T eNorm, vNorm;
	for(int i = 0; i < SUBSTEPS; i++){
		// get the goal and compute the error norm
		loadFig8Goal<T>(&goal[0],goalTime,totalTime);	evNorm<T>(currX,goal,&eNorm,&vNorm);	totalError += eNorm;
		// then get controls
		int err = getHardwareControls<T>(&(qdes[0]), 	&(udes[0]), 		tvars->x, 	tvars->u, 	tvars->KT, 	t0, 
                                         &(currX[0]),   &(currX[NUM_POS]),  tk, 
                                         tvars->ld_x, 	tvars->ld_u, 		tvars->ld_KT);
		if(err){printf("CRITICAL FAILURE ERROR ABORT MISSION\n");return 0;}
		// apply them
		_integrator<double>(&(nextX[0]),&(currX[0]),&(udes[0]),&(qdd[0]),&(I[0]),&(Tbody[0]),dt);
		for(int j=0; j < STATE_SIZE; j++){currX[j] = nextX[j];}		tk += dt_us;
	}
	#pragma unroll
	for(int j=0; j < STATE_SIZE; j++){xActual[j] = (T)currX[j];}
	loadFig8Goal<T>(&goal[0],goalTime,totalTime);	evNorm<T>(currX,goal,&eNorm,&vNorm);	totalError += eNorm;
	return (totalError / static_cast<T>(SUBSTEPS));
}
template <typename T>
__host__
void testMPC_lockstep(trajVars<T> *tvars, algTrace<T> *data, matDimms *dimms, char hardware, int doFig8){
	// define the requirements for "conversion" to the first goal
	T eNormLim = 0.05;	 T vNormLim = 0.05;	
	// define local variables
	double goalTime = 0; int initial_convergence_flag = 0; T eNorm, vNorm; T error = 0; int counter = 0; struct timeval start, end;
	// get the max iters per solve
	int itersToDo = getMaxIters(1000, 1);
	// get the max iters per solve
	int timeLimit = getTimeBudget(1000, 1); //note in ms
	// get the total time for the trajectory
	double totalTime_us = 1000000.0*static_cast<double>(getTrajTime(100, 1));	double timePrint = 0;
	// init the Ts
	tvars->t0_plant = 0; tvars->t0_sys = 0;	int64_t tActual_plant = 0; int64_t tActual_sys = 0;
	if (hardware == 'G'){
		GPUVars<T> *algvars = new GPUVars<T>;
		allocateMemory_GPU_MPC<T>(algvars, dimms, tvars);
		// load in inital trajectory and goal
		loadTraj<T>(tvars, dimms);		loadFig8Goal<T>(algvars,goalTime,totalTime_us);
		for (int i = 0; i < NUM_ALPHA; i++){
			gpuErrchk(hipMemcpy(algvars->h_d_x[i], tvars->x, (dimms->ld_x)*NUM_TIME_STEPS*sizeof(T), hipMemcpyHostToDevice));
			gpuErrchk(hipMemcpy(algvars->h_d_u[i], tvars->u, (dimms->ld_u)*NUM_TIME_STEPS*sizeof(T), hipMemcpyHostToDevice));
		}
		memcpy(algvars->xActual, tvars->x, STATE_SIZE*sizeof(T));
		// note run to conversion with no time or iter limits
		runiLQR_MPC_GPU<T>(tvars,algvars,dimms,data,tActual_sys,tActual_plant,1);
		// then start a loop of run a couple steps simulate for X steps and repeat
		while(1){
			counter++;
			gettimeofday(&start,NULL);
			runiLQR_MPC_GPU<T>(tvars,algvars,dimms,data,tActual_sys,tActual_plant,0,itersToDo,timeLimit);
			gettimeofday(&end,NULL);
			double elapsedTime_us = time_delta_us(start,end);
			tvars->t0_plant = 0; 	tActual_plant = static_cast<int>(std::floor(elapsedTime_us));
			tvars->t0_sys = 0; 		tActual_sys = static_cast<int>(std::floor(elapsedTime_us));
      		error += simulateForward<T,150>(tvars,algvars->xActual,elapsedTime_us,goalTime,totalTime_us);
			// print where are we ending up and eePos
				// int timeStepsTaken = get_steps_us_f(elapsedTime_us);
				// printf("[%d] With last successful at [%d]\nSim of %.4f is %d steps goes to:\n",counter,tvars->last_successful_solve,elapsedTime_us,timeStepsTaken);
				// printMat<T,1,STATE_SIZE>(algvars->xActual,1);
				// printf(" With expected:\n");
				// printMat<T,1,STATE_SIZE>(tvars->x + timeStepsTaken*(dimms->ld_x),1);
      		// print the state we sim to
				T *xk = &(algvars->xActual[0]);
				printf("%f,%f,%f,%f,%f,%f,%f,%f\n",timePrint,xk[0],xk[1],xk[2],xk[3],xk[4],xk[5],xk[6]);
				timePrint += elapsedTime_us;
			// print the error and the end effector position
				// T eePos[NUM_POS];   compute_eePos_scratch<T>(&(algvars->xActual[0]), &eePos[0]);
				// evNorm(algvars->xActual, algvars->xGoal, &eNorm, &vNorm);
				// printf("[[%f,%f,%f],[%f,%f,%f],%f,%f,%f],\n",eePos[0],eePos[1],eePos[2],algvars->xGoal[0],algvars->xGoal[1],algvars->xGoal[2],eNorm,error/counter,vNorm);
   			if (initial_convergence_flag){goalTime += elapsedTime_us;}
   			if(loadFig8Goal<T>(algvars,goalTime,totalTime_us) > 1){break;};
   			if (doFig8 && eNorm < eNormLim && vNorm < vNormLim){initial_convergence_flag = 1; error = 0; counter = 0;}
		}
		printf("\n\nAverage tracking error: [%f]\n",(error/counter));
		printAllTimingStats(data);
		freeMemory_GPU_MPC<T>(algvars);	delete algvars;
	}
	else{
		// TODO: update and finish this to mirror above
		// CPUVars<T> *algvars = new CPUVars<T>;
		// allocateMemory_CPU_MPC<T>(algvars, dimms, tvars);
		// // load in inital trajectory and goal
		// loadTraj<T>(tvars, dimms);		loadFig8Goal<T>(algvars,goalTime,totalTime_us);
		// memcpy(algvars->x, tvars->x, (dimms->ld_x)*NUM_TIME_STEPS*sizeof(T));
		// memcpy(algvars->u, tvars->u, (dimms->ld_u)*NUM_TIME_STEPS*sizeof(T));
		// memcpy(algvars->xActual, tvars->x, STATE_SIZE*sizeof(T));
		// // note run to conversion with no time or iter limits
		// runiLQR_MPC_CPU<T>(tvars,algvars,dimms,data,tActual_sys,tActual_plant,1);
		// // then start a loop of run a couple steps simulate for X steps and repeat
		// while(1){
		// 	gettimeofday(&start,NULL);
		// 	runiLQR_MPC_CPU<T>(tvars,algvars,dimms,data,tActual_sys,tActual_plant,0,itersToDo,timeLimit);
		// 	gettimeofday(&end,NULL);
		// 	double elapsedTime_us = 2*TIME_STEP_LENGTH_IN_us; // time_delta_us(start,end);
		// 	tvars->t0_plant = 0; 	tActual_plant = static_cast<int>(std::floor(elapsedTime_us));
		// 	tvars->t0_sys = 0; 		tActual_sys = static_cast<int>(std::floor(elapsedTime_us));
		//  error += simulateForward<T,150>(tvars,algvars->xActual,elapsedTime_us,goalTime,totalTime_us);
		// 	// print where are we ending up and eePos
		// 	int timeStepsTaken = get_steps_us_f(elapsedTime_us);
		// 	printf("[%d] Sim of %.4f is %d steps goes to:\n",counter,elapsedTime_us,timeStepsTaken);
		// 	printMat<T,1,STATE_SIZE>(algvars->xActual,1);
		// 	printf(" With expected:\n");
		// 	printMat<T,1,STATE_SIZE>(tvars->x + timeStepsTaken*(dimms->ld_x),1);
		// 	T eePos[NUM_POS];   compute_eePos_scratch<T>(&(algvars->xActual[0]), &eePos[0]);
		// 	evNorm(algvars->xActual, algvars->xGoal, &eNorm, &vNorm);
		// 	printf("[[%f,%f,%f],[%f,%f,%f],%f,%f],\n",eePos[0],eePos[1],eePos[2],algvars->xGoal[0],algvars->xGoal[1],algvars->xGoal[2],eNorm,vNorm);
		//  counter++;
		//  // if (counter > 500){printAllTimingStats(data);break;}
		//  if (doFig8 && eNorm < eNormLim && vNorm < vNormLim){initial_convergence_flag = 1; error = 0;}
		//  if (initial_convergence_flag){goalTime += elapsedTime_us;}
		//  if(loadFig8Goal<T>(algvars,goalTime,totalTime_us) > 0){break;};
		// }
		// printf("\n\nAverage tracking error: [%f]\n",(error/counter));
		// printAllTimingStats(data);
		// freeMemory_CPU_MPC<T>(algvars);	delete algvars;
	}
}
// TODO: LCM work in progress -- currently unstable
// template <typename T>
// __host__
// void testMPC_lockstepLCM(lcm::LCM *lcm_ptr, trajVars<T> *tvars, algTrace<T> *atrace, matDimms *dimms, char hardware){
// 	printf("Make sure status printer is running in another thread!\n");
// 	// get the max iters per solve
// 	int itersToDo = getMaxIters(1000, 1);
// 	// get the max iters per solve
// 	int timeLimit = getTimeBudget(1000, 1); //note in ms
// 	// get the total time for the trajectory
// 	double totalTime_us = 1000000.0*static_cast<double>(getTrajTime(100, 1));
// 	// init the Ts
// 	tvars->t0_plant = 0; tvars->t0_sys = 0;	int64_t tActual_plant = 0; int64_t tActual_sys = 0;
//     // allocate memory and construct the appropriate handlers and launch the threads
//     std::thread mpcThread;                  lcm::Subscription *mpcSub = nullptr;    lcm::Subscription *trajSub = nullptr;   // pass in sub objects so we can unsubscribe later
//     CPUVars<T> *cvars = new CPUVars<T>;     LCM_MPCLoop_Handler<T> chandler = LCM_MPCLoop_Handler<T>(cvars,tvars,dimms,atrace,itersToDo,timeLimit);
//     GPUVars<T> *gvars = new GPUVars<T>;     LCM_MPCLoop_Handler<T> ghandler = LCM_MPCLoop_Handler<T>(gvars,tvars,dimms,atrace,itersToDo,timeLimit);
//     if (hardware == 'G'){
// 		allocateMemory_GPU_MPC<T>(gvars, dimms, tvars);
// 		// load in inital trajectory and goal
// 		loadTraj<T>(tvars, dimms);		loadFig8Goal<T>(gvars,0,totalTime_us);
// 		for (int i = 0; i < NUM_ALPHA; i++){
// 			gpuErrchk(hipMemcpy(gvars->h_d_x[i], tvars->x, (dimms->ld_x)*NUM_TIME_STEPS*sizeof(T), hipMemcpyHostToDevice));
// 			gpuErrchk(hipMemcpy(gvars->h_d_u[i], tvars->u, (dimms->ld_u)*NUM_TIME_STEPS*sizeof(T), hipMemcpyHostToDevice));
// 		}
// 		memcpy(gvars->xActual, tvars->x, STATE_SIZE*sizeof(T));
// 		// note run to conversion with no time or iter limits
// 		runiLQR_MPC_GPU<T>(tvars,gvars,dimms,atrace,tActual_sys,tActual_plant,1);
// 		// then launch the MPC thread
//      	mpcThread  = std::thread(&runMPCHandler<T>, lcm_ptr, &ghandler, mpcSub);    
//     }
//     else{printf("CPU not done yet\n");}
//     lcm::LCM lcm_ptr2;	if(!lcm_ptr2.good()){printf("LCM Failed to Init\n");}
//     // launch the trajRunner
//     std::thread trajThread = std::thread(&runTrajRunner<T>, &lcm_ptr2, tvars, trajSub);
//     // launch the simulator
//     lcm::LCM lcm_ptr3;	if(!lcm_ptr3.good()){printf("LCM Failed to Init\n");}	lcm::Subscription *simSub = nullptr;
// 	double xInit[STATE_SIZE]; for(int i = 0; i < STATE_SIZE; i++){xInit[i] = 0.0;}
//     std::thread simThread = std::thread(&runSimulator, &lcm_ptr3, simSub, xInit, 1000);
//     // finally kick it all off with a status publish
//     drake::lcmt_iiwa_status dataOut; 								dataOut.utime = 0;
//     dataOut.num_joints = static_cast<int32_t>(NUM_POS); 			dataOut.joint_position_measured.resize(dataOut.num_joints);      
//     dataOut.joint_velocity_estimated.resize(dataOut.num_joints);	dataOut.joint_position_commanded.resize(dataOut.num_joints);  
//     dataOut.joint_position_ipo.resize(dataOut.num_joints);  		dataOut.joint_torque_measured.resize(dataOut.num_joints);  
//     dataOut.joint_torque_commanded.resize(dataOut.num_joints);		dataOut.joint_torque_external.resize(dataOut.num_joints);
//     for(int i = 0; i < NUM_POS; i++){
//         dataOut.joint_position_measured[i] = 0;		dataOut.joint_velocity_estimated[i] = 0;
//         dataOut.joint_position_commanded[i] = 0;  	dataOut.joint_position_ipo[i] = 0;
// 		dataOut.joint_torque_measured[i] = 0;       dataOut.joint_torque_commanded[i] = 0;		dataOut.joint_torque_external[i] = 0;
//     }
//     keyboardHold(); 		lcm_ptr->publish(ARM_STATUS_CHANNEL,&dataOut);
//     mpcThread.join();       trajThread.join();			simThread.join(); // is this what I want? to wait on all of them?
// }
// template <typename T>
// __host__
// void testMPC_LCM_singleGoal(lcm::LCM *lcm_ptr, trajVars<T> *tvars, algTrace<T> *atrace, matDimms *dimms, char hardware){
//     // launch the simulator
//     printf("Make sure the simulator is launched in another process!!!\n");
// 	// get the max iters per solve
// 	int itersToDo = getMaxIters(1000, 1);
// 	// get the max iters per solve
// 	int timeLimit = getTimeBudget(1000, 1); //note in ms
// 	// get the total time for the trajectory
// 	double totalTime_us = 1000000.0*static_cast<double>(getTrajTime(100, 1));
// 	// init the Ts
// 	tvars->t0_plant = 0; tvars->t0_sys = 0;	int64_t tActual_plant = 0; int64_t tActual_sys = 0;
//     // allocate memory and construct the appropriate handlers and launch the threads
//     std::thread mpcThread;                  lcm::Subscription *mpcSub = nullptr;    lcm::Subscription *trajSub = nullptr;   // pass in sub objects so we can unsubscribe later
//     CPUVars<T> *cvars = new CPUVars<T>;     LCM_MPCLoop_Handler<T> chandler = LCM_MPCLoop_Handler<T>(cvars,tvars,dimms,atrace,itersToDo,timeLimit);
//     GPUVars<T> *gvars = new GPUVars<T>;     LCM_MPCLoop_Handler<T> ghandler = LCM_MPCLoop_Handler<T>(gvars,tvars,dimms,atrace,itersToDo,timeLimit);
//     if (hardware == 'G'){
// 		allocateMemory_GPU_MPC<T>(gvars, dimms, tvars);
// 		// load in inital trajectory and goal
// 		loadTraj<T>(tvars, dimms);		loadFig8Goal<T>(gvars,0,totalTime_us);
// 		for (int i = 0; i < NUM_ALPHA; i++){
// 			gpuErrchk(hipMemcpy(gvars->h_d_x[i], tvars->x, (dimms->ld_x)*NUM_TIME_STEPS*sizeof(T), hipMemcpyHostToDevice));
// 			gpuErrchk(hipMemcpy(gvars->h_d_u[i], tvars->u, (dimms->ld_u)*NUM_TIME_STEPS*sizeof(T), hipMemcpyHostToDevice));
// 		}
// 		memcpy(gvars->xActual, tvars->x, STATE_SIZE*sizeof(T));
// 		// note run to conversion with no time or iter limits
// 		runiLQR_MPC_GPU<T>(tvars,gvars,dimms,atrace,tActual_sys,tActual_plant,1);
// 		// then launch the MPC thread
//      	mpcThread  = std::thread(&runMPCHandler<T>, lcm_ptr, &ghandler, mpcSub);    
//      	// setCPUForThread(&mpcThread, 1);
//     }
//     else{
// 		allocateMemory_CPU_MPC<T>(cvars, dimms, tvars);
// 		// load in inital trajectory and goal
// 		loadTraj<T>(tvars, dimms);		loadFig8Goal<T>(cvars,0,totalTime_us);
// 		memcpy(cvars->x, tvars->x, (dimms->ld_x)*NUM_TIME_STEPS*sizeof(T));
// 		memcpy(cvars->u, tvars->u, (dimms->ld_u)*NUM_TIME_STEPS*sizeof(T));
// 		memcpy(cvars->xActual, tvars->x, STATE_SIZE*sizeof(T));
// 		// note run to conversion with no time or iter limits
// 		runiLQR_MPC_CPU<T>(tvars,cvars,dimms,atrace,tActual_sys,tActual_plant,1);
// 		// then launch the MPC thread
//      	mpcThread = std::thread(&runMPCHandler<T>, lcm_ptr, &chandler, mpcSub);    
//      	// setCPUForThread(&mpcThread, 1);
//     }
//     lcm::LCM lcm_ptr2;	if(!lcm_ptr2.good()){printf("LCM Failed to Init\n");}
//     // finally launch the trajRunner
//     std::thread trajThread = std::thread(&runTrajRunner<T>, &lcm_ptr2, tvars, trajSub);
//     // setCPUForThread(&trajThread, 2);
//     printf("Both threads launched -- check simulator output!\n");
//     // clear it all 10 seconds later
//     // struct timeval start, end;       gettimeofday(&start,NULL);          while(1){gettimeofday(&end,NULL);       if (time_delta_ms(start,end) >= 10000){break;}}
//     // printf("Time up unsubscribing and joining\n");
//     // lcm_ptr->unsubscribe(mpcSub);    lcm_ptr->unsubscribe(trajSub);      // need to unsubscribe before freeing or will segfault
//     //mpcThread.join();               
//     trajThread.join();
//     // printf("Threads Joined\n");
//     // if (hardware == 'G'){freeMemory_GPU_MPC<T>(gvars);}  else{freeMemory_CPU_MPC<T>(cvars);}     delete gvars;   delete cvars;
//     // printf("memory freed\n");
// }
int main(int argc, char *argv[])
{
	srand(time(NULL));
	// test based on command line args
	char hardware = '?'; // require user input
	if (argc > 1){hardware = argv[1][0];}
	trajVars<algType> *tvars = new trajVars<algType>;	algTrace<algType> *atrace = new algTrace<algType>;	matDimms *dimms = new matDimms;
	if (hardware == 'C' || hardware == 'G'){
		int flag = atoi(&argv[1][1]);
		if (flag != 0 && flag != 1){printf("%s",errMsg); return 1;};
		testMPC_lockstep<algType>(tvars,atrace,dimms,hardware,flag);
	}
	// TODO: LCM current unstable need to update and test
	// else if (hardware == 'L'){
	// 	hardware = argv[1][1];
	// 	lcm::LCM lcm_ptr;	if(!lcm_ptr.good()){printf("LCM Failed to Init\n"); return 1;}
	// 	// testMPC_LCM_singleGoal<algType>(&lcm_ptr,tvars,atrace,dimms,hardware);
	// 	 testMPC_lockstepLCM<algType>(&lcm_ptr,tvars,atrace,dimms,hardware);
	// }
	// else if (hardware == 'P'){
	// 	char type = argv[1][1];		lcm::Subscription *sub = nullptr;
	// 	lcm::LCM lcm_ptr;	if(!lcm_ptr.good()){printf("LCM Failed to Init\n"); return 1;} 
	// 	LCM_IIWA_STATUS_printer<algType> *shandler = new LCM_IIWA_STATUS_printer<algType>;
	// 	LCM_IIWA_COMMAND_printer<algType> *chandler = new LCM_IIWA_COMMAND_printer<algType>;
	// 	if(type == 'S'){run_IIWA_STATUS_printer<algType>(&lcm_ptr,shandler,sub);}
	// 	if(type == 'C'){run_IIWA_COMMAND_printer<algType>(&lcm_ptr,chandler,sub);}
	// 	delete shandler;	delete chandler;
	// }
	else{printf("%s",errMsg); hardware = '?';}
	// free the trajVars and the wrappers
	freeTrajVars<algType>(tvars);	delete atrace;	delete tvars;	delete dimms;
	return (hardware == '?');
}